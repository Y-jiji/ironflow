
#include <hip/hip_runtime.h>
extern "C"
__global__ void sub_f32(
    const float* x,
    const float* y,
          float* z,
          int    len
) {
    for (
        int i = blockDim.x * blockIdx.x + threadIdx.x; 
            i < len;
            i += gridDim.x * blockDim.x
    ) {
        z[i] = x[i] - y[i];
    }
}

extern "C"
__global__ void sub_f64(
    const double* x,
    const double* y,
          double* z,
          int    len
) {
    for (
        int i = blockDim.x * blockIdx.x + threadIdx.x; 
            i < len;
            i += gridDim.x * blockDim.x
    ) {
        z[i] = x[i] - y[i];
    }
}

extern "C"
__global__ void sub_i32(
    const int* x,
    const int* y,
          int* z,
          int  len
) {
    for (
        int i = blockDim.x * blockIdx.x + threadIdx.x; 
            i < len;
            i += gridDim.x * blockDim.x
    ) {
        z[i] = x[i] - y[i];
    }
}

extern "C"
__global__ void sub_i64(
    const long* x,
    const long* y,
          long* z,
          int   len
) {
    for (
        int i = blockDim.x * blockIdx.x + threadIdx.x; 
            i < len;
            i += gridDim.x * blockDim.x
    ) {
        z[i] = x[i] - y[i];
    }
}